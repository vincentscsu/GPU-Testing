#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "check.h"

/*
 * This program generates two arrays of 16M elements to test the device's
 * cpability.
 */

//Host summation funciton
void sum_CPU(float *A, float *B, float *C, int n)
{
    int i;

    for (i = 0;i < n; i++)
    {
        C[i] = A[i] + B[i];
    }
}

//GPU summation kernel
__global__ void sum_GPU(float *A, float *B, float *C, int n)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    C[i] = A[i] + B[i];
}

//random data generation
void data_init(float *data, int size)
{   
    //seed for random generation
    time_t t;
    srand((unsigned) time(&t));

    for (int i = 0; i < size; i++)
    {
        //mask off to get only lower two bytes
        data[i] = (float)(rand() & 0xFF);
    }
}

//check results
void check_result(float *A, float *B, int size)
{
    double eps = 1.0e-8;
    int i;
    int flag = 1;

    for (i = 0; i < size; i++)
    {
        if (abs(A[i] - B[i]) > eps)
        {   
            flag = 0;
            break;
        }
    }
    if (flag == 1)
    {
        printf("Results match!\n");
    }
    else
    {
        printf("Results from CPU and GPU do not match at %d (%f, %f)\n",
                i, A[i], B[i]);
    }
}


int main()
{
    //return device info
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    //initialize data size;
    int size = 1 << 24;
    printf("Array size: %d elements\n", size);
    
    //allocate host memory
    float *h_A, *h_B, *h_C, *cpu_result;
    h_A = (float*)malloc(size*sizeof(float));
    h_B = (float*)malloc(size*sizeof(float));
    h_C = (float*)malloc(size*sizeof(float));
    cpu_result = (float*)malloc(size*sizeof(float));

    //initialize data on host
    data_init(h_A, size);
    data_init(h_B, size);
    memset(h_C, 0, size);
    memset(cpu_result, 0 ,size);

    //get ready to time cpu summation
    double start, elapsed;

    //summation on host for reference
    start = seconds();
    sum_CPU(h_A, h_B, cpu_result, size);
    elapsed = seconds() - start;
    printf("sum_CPU uesed: %f sec\n", elapsed);
    
    //allocate device global memory
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float**)&d_A, size*sizeof(float)));
    CHECK(hipMalloc((float**)&d_B, size*sizeof(float)));
    CHECK(hipMalloc((float**)&d_C, size*sizeof(float)));

    //copy data from host to device
    CHECK(hipMemcpy(d_A, h_A, size*sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, size*sizeof(float), hipMemcpyHostToDevice));

    //kernel specs
    dim3 block (1024,1,1);
    dim3 grid (((size-1)/block.x)+1, 1 ,1);
    
    //get ready to time the kernel
    start = seconds();

    //launch kernel
    sum_GPU<<<grid,block>>>(d_A, d_B, d_C, size);
    CHECK(hipDeviceSynchronize());
    elapsed = seconds() - start;
    printf("sum_GPU<<< %d , %d >>> used: %f sec\n", grid.x, block.x, elapsed);

    //check kernel errors
    CHECK(hipGetLastError());

    //copy result from device to host
    CHECK(hipMemcpy(h_C, d_C, size*sizeof(float), hipMemcpyDeviceToHost));

    //check results
    check_result(cpu_result, h_C, size);

    //free device global memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    //free host memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(cpu_result);

    return 0;
}

